#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "specialized_api.h"
#include "complex.cuh"
#include "math_fn.cuh"
#include "iterators.cuh"
#include "tensor.cuh"

//==============================================================================
// Swift importable C interface functions
//==============================================================================

//==============================================================================
// Julia Set

// tensorA Element
template<typename IterA, typename IterO>
__global__ void mapJulia(
    IterA iterA,
    IterO iterO,
    const typename IterO::T::RealType tolerance2,
    const typename IterO::T c,
    int iterations
) {
    // 0.000416s
    const auto p = typename IterO::Logical(blockIdx, blockDim, threadIdx);
    if (iterO.isInBounds(p)) {
        auto z = iterA[p];
        float d = iterations;
        for (int j = 0; j < iterations; ++j) {
            z = z * z + c;
            if (abs2(z) > tolerance2) {
                d = min(d, float(j));
                break;
            }
        }
        iterO[p] = d;
    }
}

template<typename A>
hipError_t juliaFlat(
    const void* pA,
    const void* pConstant,
    const void* pTolerance,
    size_t iterations,
    size_t count,
    void* pOut,
    hipStream_t stream
) {
    typedef typename A::RealType RT;

    const A* a = static_cast<const A*>(pA);
    A* out = static_cast<A*>(pOut);
    const RT tolerance = *static_cast<const RT*>(pTolerance);
    const A c = *static_cast<const A*>(pConstant);

    auto iterA = Flat(a, count);
    auto iterO = Flat(out, count);
    auto tolerance2 = RT(float(tolerance) * float(tolerance));

    dim3 tile = tileSize(iterO.count);
    dim3 grid = gridSize(iterO.count, tile);

    mapJulia<<<grid, tile, 0, stream>>>(iterA, iterO, tolerance2, c, iterations);
    return hipSuccess;
}

hipError_t srtJuliaFlat(
    srtDataType type,
    const void* a,
    const void* constant,
    const void* tolerance,
    size_t iterations,
    size_t count,
    void* out,
    hipStream_t stream
) {
    switch (type) {
    case complex16F: return juliaFlat<Complex<float16>>(a, constant, tolerance, iterations, count, out, stream);
    case complex32F: return juliaFlat<Complex<float>>(a, constant, tolerance, iterations, count, out, stream);
    default: return hipErrorNotSupported;
    }
}

//==============================================================================
// Mandelbrot Set

// tensorA Element
template<typename IterA, typename IterO>
__global__ void mapMandelbrot(
    IterA iterA,
    IterO iterO,
    const float tolerance2,
    int iterations
) {
    // 0.00111s
    const auto p = typename IterO::Logical(blockIdx, blockDim, threadIdx);
    if (iterO.isInBounds(p)) {
        const Complex<float> x = iterA[p];
        Complex<float> z = x;
        float d = iterations;
        for (int j = 1; j < iterations; ++j) {
            z = z * z + x;
            if (abs2(z) > tolerance2) {
                d = min(d, float(j));
                break;
            }
        }
        iterO[p] = d;
    }
}

hipError_t srtMandelbrotFlat(
    srtDataType type,
    const void* pA,
    const void* pTolerance,
    size_t iterations,
    size_t count,
    void* pOut,
    hipStream_t stream
) {
    assert(type == complex32F);
    const Complex<float>* a = static_cast<const Complex<float>*>(pA);
    float* out = static_cast<float*>(pOut);
    const float tolerance = *static_cast<const float*>(pTolerance);

    auto iterA = Flat(a, count);
    auto iterO = Flat(out, count);
    float tolerance2 = tolerance * tolerance;

    dim3 tile = tileSize(iterO.count);
    dim3 grid = gridSize(iterO.count, tile);

    mapMandelbrot<<<grid, tile, 0, stream>>>(iterA, iterO, tolerance2, iterations);
    return hipSuccess;
}

