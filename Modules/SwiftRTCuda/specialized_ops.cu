#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "specialized_api.cuh"
#include "complex.cuh"
#include "iterators.cuh"

//==============================================================================
// Swift importable C interface functions
//==============================================================================

//------------------------------------------------------------------------------
// tensorA Element
template<typename IterA, typename IterO>
__global__ void mapJulia(
    IterA iterA,
    IterO iterO,
    const float tolerance,
    const Complex<float> C,
    int iterations
) {
    // 0.000790s
    const auto p = IterO::Logical(blockIdx, blockDim, threadIdx);
    if (iterO.isInBounds(p)) {
        float t2 = tolerance * tolerance;
        Complex<float> Z = iterA[p];
        auto index = iterO.linear(p);
        float d = iterO[index];
        for (int j = 0; j < iterations; ++j) {
            Z = Z * Z + C;
            if (abs2(Z) > t2) {
                d = min(d, float(j));
                break;
            }
        }
        iterO[index] = d;
    }
}


hipError_t srtJulia(
    const void* pz, const srtTensorDescriptor* pzDesc,
    void* pdivergence, const srtTensorDescriptor* pdDesc,
    const void* ptolerance,
    const void* pC,
    size_t iterations,
    hipStream_t stream
) {
    const TensorDescriptor& zDesc = static_cast<const TensorDescriptor&>(*pzDesc);
    const TensorDescriptor& dDesc = static_cast<const TensorDescriptor&>(*pdDesc);

    const Complex<float>* z = static_cast<const Complex<float>*>(pz);
    float* d = static_cast<float*>(pdivergence);
    const float tolerance = *static_cast<const float*>(ptolerance);
    const Complex<float> C = *static_cast<const Complex<float>*>(pC);

    auto iterZ = Flat(z, zDesc.count);
    auto iterD = Flat(d, dDesc.count);

    dim3 tile = tileSize(iterD.count);
    dim3 grid = gridSize(iterD.count, tile);

    mapJulia<<<grid, tile, 0, stream>>>(iterZ, iterD, tolerance, C, iterations);
    return hipSuccess;
}
