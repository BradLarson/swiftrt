//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "srt_dispatch.h"
#include "compare_fn.h"
#include "compare_vjp.h"

//==============================================================================
// Swift importable C interface functions
//==============================================================================

//------------------------------------------------------------------------------
Op2(And, andElements, (isBool<T>() && isBool<Out>()))

hipError_t srtAnd(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<And>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op3(AlmostEqual, almostEqual, (isFloating<T>() && isBool<Out>()))

hipError_t srtElementsAlmostEqual(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* tolerance,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<AlmostEqual>(a, aDesc, b, bDesc, tolerance, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Equal, equal, (isEquatable<T>() && isBool<Out>()))

hipError_t srtEqual(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Equal>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Greater, greater, (isComparable<T>() && isBool<Out>()))

hipError_t srtGreater(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Greater>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtGreaterTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Greater>(a, aDesc, element, out, oDesc, stream);
}
    
//------------------------------------------------------------------------------
Op2(GreaterOrEqual, greaterOrEqual, (isComparable<T>() && isBool<Out>()))

hipError_t srtGreaterOrEqual(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<GreaterOrEqual>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtGreaterOrEqualTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<GreaterOrEqual>(a, aDesc, element, out, oDesc, stream);
}
    
//------------------------------------------------------------------------------
Op2(Less, less, (isComparable<T>() && isBool<Out>()))

hipError_t srtLess(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Less>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtLessTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Less>(a, aDesc, element, out, oDesc, stream);
}
    
//------------------------------------------------------------------------------
Op2(LessOrEqual, lessOrEqual, (isComparable<T>() && isBool<Out>()))

hipError_t srtLessOrEqual(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<LessOrEqual>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtLessOrEqualTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<LessOrEqual>(a, aDesc, element, out, oDesc, stream);
}
    
//------------------------------------------------------------------------------
Op2(MinElements, minElements, (isComparable<T>() && isSame<T, Out>()))

hipError_t srtMin(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<MinElements>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtMinTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<MinElements>(a, aDesc, element, out, oDesc, stream);
}
    
//------------------------------------------------------------------------------
Op2(MaxElements, maxElements, (isComparable<T>() && isSame<T, Out>()))

hipError_t srtMax(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<MaxElements>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtMaxTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<MaxElements>(a, aDesc, element, out, oDesc, stream);
}
    
//------------------------------------------------------------------------------
Op2(NotEqualElements, notEqual, (isEquatable<T>() && isBool<Out>()))

hipError_t srtNotEqual(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<NotEqualElements>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Or, orElements, (isBool<T>() && isBool<Out>()))

hipError_t srtOr(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Or>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
OpTTU(Replace, conditionalAssign, (isSame<T,Out>() && isBool<U>()))

hipError_t srtReplace(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* condition, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsABC(paDesc, pbDesc, pcDesc, poDesc)
    return selectTTU<Replace>(b, bDesc, a, aDesc, condition, cDesc, out, oDesc, stream);
}

//==============================================================================
OpSame3(VjpMin, vjpMin, (isComparable<T>()))

hipError_t srtVjpMin(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsABC(paDesc, pbDesc, pcDesc, poDesc)
    return select<VjpMin>(a, aDesc, b, bDesc, c, cDesc, out, oDesc, stream);
}

hipError_t srtVjpMinTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

OpSame32(VjpMin32, vjpMax, (isComparable<T>()))

hipError_t srtVjpMinOO(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* outT, const srtTensorDescriptor* oTDesc,
    void* outF, const srtTensorDescriptor* oFDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtVjpMinTEOO(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* outT, const srtTensorDescriptor* oTDesc,
    void* outF, const srtTensorDescriptor* oFDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

//==============================================================================
OpSame3(VjpMax, vjpMax, (isComparable<T>()))

hipError_t srtVjpMax(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsABC(paDesc, pbDesc, pcDesc, poDesc)
    return select<VjpMax>(a, aDesc, b, bDesc, c, cDesc, out, oDesc, stream);
}

hipError_t srtVjpMaxTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

OpSame32(VjpMax32, vjpMax, (isComparable<T>()))

hipError_t srtVjpMaxOO(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* outT, const srtTensorDescriptor* oTDesc,
    void* outF, const srtTensorDescriptor* oFDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtVjpMaxTEOO(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* outT, const srtTensorDescriptor* oTDesc,
    void* outF, const srtTensorDescriptor* oFDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}
