#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <bits/stdint-uintn.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "fillOps.h"
#include "dispatchHelpers.h"

//==============================================================================
// Swift importable C interface functions
//==============================================================================

hipError_t srtCopy(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

//==============================================================================
// srtFill

// kernel
template <typename E, typename IndexO>
__global__ void mapElementFill(E *out, IndexO indexO, E element) {
  auto position = IndexO::Logical(blockIdx, blockDim, threadIdx);
  if (indexO.isInBounds(position)) {
    int i = indexO.linear(position);
    out[i] = element;
  }
}

template <typename E>
static hipError_t elementFill(void *pOut, const TensorDescriptor &oDesc,
                               const E element, hipStream_t stream,
                               const int shiftCount = 0) {
  E *out = static_cast<E *>(pOut);
  int count = shiftDownRoundingUp(oDesc.count, shiftCount);
  dim3 tile = tileSize(count);
  dim3 grid = gridSize<1>(oDesc, tile);
  mapElementFill<E, Flat><<<grid, tile, 0, stream>>>(out, Flat(oDesc), element);
  return hipSuccess;
}

//------------------------------------------------------------------------------
/// srtFill
/// Fills the output buffer with the element value
///
/// - Parameters:
///  - out: pointer to device memory output buffer
///  - poDesc: pointer to output srtTensorDescriptor
///  - element: pointer to element fill value in host memory
///  - stream: the execution stream
hipError_t srtFill(
    void* out, const srtTensorDescriptor* poDesc,
    const void* element,
    hipStream_t stream
) {
    const TensorDescriptor& oDesc = static_cast<const TensorDescriptor&>(*poDesc);
    assert(oDesc.isDense());

    // The output type is converted to a packed type if possible for a faster
    // fill operation. The output buffer space is guaranteed to be rounded
    // up to a multiple of the largest packed type so we don't have to worry
    // about writing out of bounds.
    switch(oDesc.type) {
        case HIP_R_32F:
          return elementFill<float>(out, oDesc, *(float *)element, stream);
        case HIP_R_64F:
          return elementFill<double>(out, oDesc, *(double *)element, stream);

        case HIP_R_16F: 
        case HIP_R_16BF:
        case HIP_R_16I:
        case HIP_R_16U:
            // pack 16 bit elements
            return elementFill<uint32_t>(out, oDesc, fillWord<uint16_t>(element), stream, 1);

        case HIP_R_8I: 
        case HIP_R_8U:
            // pack 8 bit elements
            return elementFill<uint32_t>(out, oDesc, fillWord<uint8_t>(element), stream, 2);
        default: return hipErrorNotSupported;
    }
}

//==============================================================================
/// srtFillRange
/// Fills the output with logical position indexes  
hipError_t srtFillRange(
    void* out, const srtTensorDescriptor* oDesc,
    const long lower,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

//==============================================================================
hipError_t srtEye(
    void* out, const srtTensorDescriptor* oDesc,
    const long offset,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomUniform(
    void* out, const srtTensorDescriptor* oDesc,
    const void* lower,
    const void* upper,
    const uint64_t seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomNormal(
    void* out, const srtTensorDescriptor* oDesc,
    const void* mean,
    const void* std,
    const uint64_t seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomNormalTensorArgs(
    void* out, const srtTensorDescriptor* oDesc,
    const void* meanTensor,
    const void* stdTensor,
    const uint64_t seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomTruncatedNormal(
    void* out, const srtTensorDescriptor* oDesc,
    const void* mean,
    const void* std,
    const uint64_t seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomTruncatedNormalTensorArgs(
    void* out, const srtTensorDescriptor* oDesc,
    const void* meanTensor,
    const void* stdTensor,
    const uint64_t seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}
