//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <assert.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "fillOps.h"

//==============================================================================
// kernels
//==============================================================================

//==============================================================================
// dynamic dispatch functions
//==============================================================================


//==============================================================================
// Swift importable C interface functions
//==============================================================================

hipError_t srtCopy(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFill(
    void* out, const srtTensorDescriptor* oDesc,
    const void* element,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillWithRange(
    void* out, const srtTensorDescriptor* oDesc,
    const long lower,
    const long upper, 
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtEye(
    void* out, const srtTensorDescriptor* oDesc,
    const long offset,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomUniform(
    void* out, const srtTensorDescriptor* oDesc,
    const void* lower,
    const void* upper,
    const uint64_t* seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomNormal(
    void* out, const srtTensorDescriptor* oDesc,
    const void* mean,
    const void* std,
    const uint64_t* seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomNormalTensorArgs(
    void* out, const srtTensorDescriptor* oDesc,
    void* meanTensor,
    void* stdTensor,
    const uint64_t* seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomTruncatedNormal(
    void* out, const srtTensorDescriptor* oDesc,
    const void* mean,
    const void* std,
    const uint64_t* seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtFillRandomTruncatedNormalTensorArgs(
    void* out, const srtTensorDescriptor* oDesc,
    void* meanTensor,
    void* stdTensor,
    const uint64_t* seed,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}
