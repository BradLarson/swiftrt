#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "mathOps.h"
#include "kernelHelpers.h"
#include "index.h"

//==============================================================================
// ops
//==============================================================================

template<typename E>
struct Abs {
    __device__ inline static E op(const E& x) { return abs(x); }
};

//==============================================================================
// kernels
//==============================================================================

//==============================================================================
// dynamic dispatch functions
//==============================================================================


//==============================================================================
// Swift importable C interface functions
//==============================================================================

hipError_t srtAbs(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAcos(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAcosh(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAsin(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAsinh(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAtan(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAtan2(
    const void* y, const srtTensorDescriptor* yDesc,
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtAtanh(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtCos(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtCosh(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtErf(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtErfc(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtExp(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtExp2(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtExp10(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtExpMinusOne(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtGamma(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtHypot(
    const void* x, const srtTensorDescriptor* xDesc,
    const void* y, const srtTensorDescriptor* yDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLog(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLogOnePlus(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLog2(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLog10(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLogGamma(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtNeg(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtPow(
    const void* x, const srtTensorDescriptor* xDesc,
    const void* y, const srtTensorDescriptor* yDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtPowN(
    const void* x, const srtTensorDescriptor* xDesc, long n,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtRoot(
    const void* x, const srtTensorDescriptor* xDesc, long n,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtSigmoid(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtSign(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtSin(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtSinh(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtSqrt(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtSquared(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtTan(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtTanh(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}
