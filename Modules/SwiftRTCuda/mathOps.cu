#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "mathOps.h"
#include "kernelHelpers.h"
#include "index.h"

//==============================================================================
// ops
//==============================================================================

template<typename E>
struct Abs {
    __device__ inline static E op(const E& x) { return abs(x); }
};


hipError_t srtAbs(
    const void* x, const srtTensorDescriptor* xDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    
}
