#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "disp.h"
#include "mathOps.h"
#include "mathSupplemental.h"
#include <type_traits>

//------------------------------------------------------------------------------
// greaterElements
template<typename T>
__device__ inline bool greaterElements(const T& a, const T& b) {
    return a > b;
}


//==============================================================================
// Swift importable C interface functions
//==============================================================================

IntFloatComplexOpA(Abs, abs)

hipError_t srtAbs(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Abs>(a, aDesc, out, oDesc, stream);
}

FloatOpA(Acos, acos)

// Must be promoted types
hipError_t srtAcos(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Acos>(a, aDesc, out, oDesc, stream);
}

IntFloatComplexOpAB(Add, add)

hipError_t srtAdd(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Add>(a, aDesc, b, bDesc, out, oDesc, stream);
}

IntFloatComplexOpAB(Greater, greaterElements)

hipError_t srtGreater(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Greater>(a, aDesc, b, bDesc, out, oDesc, stream);
}
