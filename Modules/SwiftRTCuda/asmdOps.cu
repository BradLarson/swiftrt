#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <assert.h>
#include <cstddef>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

// Element wise Add, Subtract, Multiply, Divide ops
#include "asmdOps.h"
#include "kernelHelpers.h"

//==============================================================================
// #if (__CUDA_ARCH__ < 800)
// __device__ __forceinline__ __hip_bfloat162 operator+(const __hip_bfloat162& l, const __hip_bfloat162& r) {
//     __hip_bfloat162 c;
//     c.x = __float2bfloat16_rn(__bfloat162float(l.x) + __bfloat162float(r.x));
//     c.y = __float2bfloat16_rn(__bfloat162float(l.y) + __bfloat162float(r.y));
//     return c;
// }
// #endif

__device__ inline __hip_bfloat162 add(const __hip_bfloat162& l, const __hip_bfloat162& r) {
    __hip_bfloat162 c;
    c.x = __float2bfloat16_rn(__bfloat162float(l.x) + __bfloat162float(r.x));
    c.y = __float2bfloat16_rn(__bfloat162float(l.y) + __bfloat162float(r.y));
    return c;
}

// template<typename T>
// __global__ void add_bfloat162(
//     const void *va, int strideA,
//     const void *vb, int strideB,
//     void *vc,
//     unsigned count
// ) {
//     auto a = static_cast<const T*>(va);
//     auto b = static_cast<const T*>(vb);
//     auto c = static_cast<T*>(vc);

//     GRID_STRIDE_LOOP(ai, strideA, bi, strideB, ci, count) {
//         #if (__CUDA_ARCH__ >= 800)
//             c[ci] = a[ai] + b[bi];
//         #else
//             c[ci] = add(a[ai], b[bi]);
//         #endif
//     }
// }

//==============================================================================
// ops
//==============================================================================

template<typename T>
struct Add {
    __device__ __forceinline__ T operator()(const T& a, const T& b) { return a + b; }
};

template<typename T>
struct Sub {
    __device__ __forceinline__ T operator()(const T& a, const T& b) { return a - b; }
};

template<typename T>
struct Mul {
    __device__ __forceinline__ T operator()(const T& a, const T& b) { return a * b; }
};

template<typename T>
struct Div {
    __device__ __forceinline__ T operator()(const T& a, const T& b) { return a / b; }
};

//==============================================================================
// buffer iterable kernels
//==============================================================================

//------------------------------------------------------------------------------
// tensor2
template<template<typename U> class Op, typename T>
__global__ void tensor2(const T *a, const T *b, T *out, unsigned count) {
    Op<T> op;
    GRID_LOOP(i, count) {
        out[i] =  op(a[i], b[i]);
    }
}

//------------------------------------------------------------------------------
// tensorScalar
template<template<typename U> class Op, typename T>
__global__ void tensorScalar(const T *elements, const T *scalar, T *out, unsigned count) {
    Op<T> op;
    GRID_LOOP(i, count) {
        out[i] = op(elements[i], scalar[0]);
    }
}

//------------------------------------------------------------------------------
// scalarTensor
template<template<typename U> class Op, typename T>
__global__ void scalarTensor(const T *scalar, const T *elements, T *out, unsigned count) {
    Op<T> op;
    GRID_LOOP(i, count) {
        out[i] = op(scalar[0], elements[i]);
    }
}

//------------------------------------------------------------------------------
/// combine
/// invokes the correct kernel to combine the elements of the two tensors
/// handling the cases of elements and single scalar sets.
///
template<template<typename U> class Op, typename T>
static void combine(
    unsigned blocks, 
    unsigned threads,
    const void *pA, size_t strideA, 
    const void *pB, size_t strideB,
    void *pOut, size_t count,
    hipStream_t stream
) {
    const T* a = static_cast<const T*>(pA);
    const T* b = static_cast<const T*>(pB);
    T* out = static_cast<T*>(pOut);

    if (strideA == 1 && strideB == 1) {
        // combine two sets of elements
        tensor2<Op, T><<<blocks, threads, 0, stream>>>(a, b, out, count);

    } else if (strideA == 1 && strideB == 0) {
        // combine elements with a scalar
        tensorScalar<Op, T><<<blocks, threads, 0, stream>>>(a, b, out, count); 

    } else if (strideA == 0 && strideB == 1) {
        // combine a scalar with elements
        scalarTensor<Op, T><<<blocks, threads, 0, stream>>>(a, b, out, count); 
    }
}

//------------------------------------------------------------------------------
// resolveType
// this function is for dense tensors that can be flattened where
// `isBufferIterable == true`, so strides must equal 0 or 1
template<template<typename U> class Op>
static hipError_t resolveType(
    hipDataType type, 
    const void *a, size_t strideA, 
    const void *b, size_t strideB,
    void *out, size_t count,
    hipStream_t stream
) {
    // make sure sizes fit within Cuda limitations
    assert(count <= UINT32_MAX);
    assert(strideA == 0 || strideA == 1 && strideB == 0 || strideB == 1);
    KernelPreCheck(stream);

    unsigned blocks = BLOCK_COUNT(count);
    unsigned threads = THREADS_PER_BLOCK;

    switch(type) {
        case HIP_R_32F: combine<Op, float>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        // *** Figure out how to define operator+ for this type so it works in emulation mode
        // case HIP_R_16BF: {
        //     unsigned n = shiftDownRoundingUp(countC, 1);
        //     addScalar_bfloat162<__hip_bfloat162><<<BLOCK_COUNT(n), threads, 0, stream>>>(a, pScalar, c, n);
        //     break;
        // }
        case HIP_R_16F: {
            unsigned n = shiftDownRoundingUp(count, 1);
            combine<Add, __half>(BLOCK_COUNT(n), threads, a, strideA, b, strideB, out, count, stream); break;
            break;
        }
        case HIP_R_8I:  combine<Op, int8_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        case HIP_R_8U:  combine<Op, uint8_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        case HIP_R_16I: combine<Op, int16_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        case HIP_R_16U: combine<Op, uint16_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        case HIP_R_64F: combine<Op, double>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        default: printf("hipDataType not implemented"); exit(1);
    }

    return KernelPostCheck(stream);
}

//==============================================================================
// Swift importable C interface functions
//==============================================================================

//------------------------------------------------------------------------------
// srtAdd
hipError_t srtAdd(
    hipDataType type, 
    const void *a, size_t strideA, 
    const void *b, size_t strideB,
    void *out, size_t count,
    hipStream_t stream
) {
    return resolveType<Add>(type, a, strideA, b, strideB, out, count, stream);
}

//------------------------------------------------------------------------------
// srtSub
hipError_t srtSub(
    hipDataType type, 
    const void *a, size_t strideA, 
    const void *b, size_t strideB,
    void *out, size_t count,
    hipStream_t stream
) {
    return resolveType<Sub>(type, a, strideA, b, strideB, out, count, stream);
}

//------------------------------------------------------------------------------
// srtMul
hipError_t srtMul(
    hipDataType type, 
    const void *a, size_t strideA, 
    const void *b, size_t strideB,
    void *out, size_t count,
    hipStream_t stream
) {
    return resolveType<Mul>(type, a, strideA, b, strideB, out, count, stream);
}

//------------------------------------------------------------------------------
// srtDiv
hipError_t srtDiv(
    hipDataType type, 
    const void *a, size_t strideA, 
    const void *b, size_t strideB,
    void *out, size_t count,
    hipStream_t stream
) {
    return resolveType<Div>(type, a, strideA, b, strideB, out, count, stream);
}

//******************************************************************************
//******************************************************************************

//==============================================================================
// strided index kernels
//==============================================================================

//------------------------------------------------------------------------------
// strided2
template<template<typename U> class Op, typename T, size_t Rank>
__global__ void strided2(
    const T *a, const Index<Rank> aIndex,
    const T *b, const Index<Rank> bIndex,
    T *out, const Index<Rank> oIndex,
    unsigned count
) {
    Op<T> op;
    // GRID_LOOP_STRIDED(ai, strideA, bi, strideB, oi, count) {
    //     out[oi] =  op(a[ai], b[bi]);
    // }
}

//------------------------------------------------------------------------------
// resolveType
// this function is for dense tensors that can be flattened where
// `isBufferIterable == true`, so strides must equal 0 or 1
template<template<typename U> class Op>
static hipError_t resolveType(
    const void *a, const srtTensorDescriptor& aDesc,
    const void *b, const srtTensorDescriptor& bDesc,
    void *out, const srtTensorDescriptor& oDesc,
    hipStream_t stream
) {
    // make sure sizes fit within Cuda limitations
    size_t count = oDesc.count;
    assert(count <= UINT32_MAX);
    assert(aDesc.type == bDesc.type == oDesc.type);

    KernelPreCheck(stream);

    // unsigned blocks = BLOCK_COUNT(count);
    // unsigned threads = THREADS_PER_BLOCK;

    switch(aDesc.type) {
        // case HIP_R_32F: strided2<<<blocks, threads, 0, stream>>>(a, ); break;
    //     // *** Figure out how to define operator+ for this type so it works in emulation mode
    //     // case HIP_R_16BF: {
    //     //     unsigned n = shiftDownRoundingUp(countC, 1);
    //     //     addScalar_bfloat162<__hip_bfloat162><<<BLOCK_COUNT(n), threads, 0, stream>>>(a, pScalar, c, n);
    //     //     break;
    //     // }
    //     case HIP_R_16F: {
    //         unsigned n = shiftDownRoundingUp(count, 1);
    //         combine<Add, __half>(BLOCK_COUNT(n), threads, a, strideA, b, strideB, out, count, stream); break;
    //         break;
    //     }
    //     case HIP_R_8I:  combine<Op, int8_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
    //     case HIP_R_8U:  combine<Op, uint8_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
    //     case HIP_R_16I: combine<Op, int16_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
    //     case HIP_R_16U: combine<Op, uint16_t>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
    //     case HIP_R_64F: combine<Op, double>(blocks, threads, a, strideA, b, strideB, out, count, stream); break;
        default: printf("hipDataType not implemented"); exit(1);
    }

    return KernelPostCheck(stream);
}

//==============================================================================
// Swift importable C interface functions
// performs the operation with fully strided index calculations
//==============================================================================

//------------------------------------------------------------------------------
// strStridedAdd
hipError_t strStridedAdd(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    return resolveType<Add>(a, *aDesc, b, *bDesc, out, *oDesc, stream);
}

