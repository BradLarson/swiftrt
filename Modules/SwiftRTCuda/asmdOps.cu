#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <assert.h>
#include <bits/stdint-uintn.h>
#include <cstddef>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

// Element wise Add, Subtract, Multiply, Divide ops
#include "asmdOps.h"
#include "kernelHelpers.h"
#include "index.h"

//==============================================================================
// #if (__CUDA_ARCH__ < 800)
// __device__ __forceinline__ __hip_bfloat162 operator+(const __hip_bfloat162& l, const __hip_bfloat162& r) {
//     __hip_bfloat162 c;
//     c.x = __float2bfloat16_rn(__bfloat162float(l.x) + __bfloat162float(r.x));
//     c.y = __float2bfloat16_rn(__bfloat162float(l.y) + __bfloat162float(r.y));
//     return c;
// }
// #endif

__device__ inline __hip_bfloat162 add(const __hip_bfloat162& l, const __hip_bfloat162& r) {
    __hip_bfloat162 c;
    c.x = __float2bfloat16_rn(__bfloat162float(l.x) + __bfloat162float(r.x));
    c.y = __float2bfloat16_rn(__bfloat162float(l.y) + __bfloat162float(r.y));
    return c;
}

// template<typename E>
// __global__ void add_bfloat162(
//     const void *va, int strideA,
//     const void *vb, int strideB,
//     void *vc,
//     unsigned count
// ) {
//     auto a = static_cast<const E*>(va);
//     auto b = static_cast<const E*>(vb);
//     auto c = static_cast<E*>(vc);

//     GRID_STRIDE_LOOP(ai, strideA, bi, strideB, ci, count) {
//         #if (__CUDA_ARCH__ >= 800)
//             c[ci] = a[ai] + b[bi];
//         #else
//             c[ci] = add(a[ai], b[bi]);
//         #endif
//     }
// }

//==============================================================================
// ops
//==============================================================================

template<typename E>
struct Add {
    __device__ __forceinline__ E operator()(const E& a, const E& b) { return a + b; }
};

template<typename E>
struct Sub {
    __device__ __forceinline__ E operator()(const E& a, const E& b) { return a - b; }
};

template<typename E>
struct Mul {
    __device__ __forceinline__ E operator()(const E& a, const E& b) { return a * b; }
};

template<typename E>
struct Div {
    __device__ __forceinline__ E operator()(const E& a, const E& b) { return a / b; }
};

//==============================================================================
// kernels
template<template<typename U> class Op, typename E>
__global__ void abSingleSingle(const E *a, const E *b, E *out, uint32_t count) 
{
    Op<E> op;
    E element = op(a[0], b[0]);
    GRID_LOOP(i, count) {
        out[i] = element;
    }
}

template<template<typename U> class Op, typename E>
__global__ void abFlatSingle(const E *a, const E *b, E *out, uint32_t count) 
{
    Op<E> op;
    GRID_LOOP(i, count) {
        out[i] = op(a[i], b[0]);
    }
}

template<template<typename U> class Op, typename E>
__global__ void abSingleFlat(const E *a, const E *b, E *out, uint32_t count) 
{
    Op<E> op;
    GRID_LOOP(i, count) {
        out[i] = op(a[0], b[i]);
    }
}

template<template<typename U> class Op, typename E>
__global__ void abFlatFlat(const E *a, const E *b, E *out, uint32_t count) 
{
    Op<E> op;
    GRID_LOOP(i, count) {
        out[i] = op(a[i], b[i]);
    }
}

//------------------------------------------------------------------------------
/// combine
/// invokes the correct kernel to combine the elements of the two tensors
/// handling the cases of elements and single single sets.
///
template<template<typename U> class Op, int R, typename E>
static void combine(
    const void* pA, const srtTensorDescriptor* paDesc,
    const void* pB, const srtTensorDescriptor* pbDesc,
    void* pOut, const srtTensorDescriptor* poDesc,
    hipStream_t stream,
    unsigned shiftCount = 0 
) {
    // statically cast types from C interface to use with c++ templates
    E* out = static_cast<E*>(pOut);
    const E* a = static_cast<const E*>(pA);
    const E* b = static_cast<const E*>(pB);
    const TensorDescriptor& oDesc = static_cast<const TensorDescriptor&>(*poDesc);
    const TensorDescriptor& aDesc = static_cast<const TensorDescriptor&>(*paDesc);
    const TensorDescriptor& bDesc = static_cast<const TensorDescriptor&>(*pbDesc);

    // the count is divided in cases where values are handled as short vectors
    unsigned count = shiftDownRoundingUp(oDesc.count, shiftCount);

    // make sure total count fits within Cuda limitations
    assert(count <= UINT32_MAX);
    unsigned blocks = BLOCK_COUNT(count);
    unsigned threads = THREADS_PER_BLOCK;
    
    if (bDesc.isSingle()) {
        if (aDesc.isSingle()) {
            // single op single --> dense
            abSingleSingle<Op,E><<<blocks, threads, 0, stream>>>(a, b, out, count);

        } else if (aDesc.isDense()) {
            // dense op single --> dense
            abFlatSingle<Op,E><<<blocks, threads, 0, stream>>>(a, b, out, count);

        } else {
            // strided op single --> dense
        }
    } else if (bDesc.isDense()) {
        if (aDesc.isSingle()) {
            // single op dense --> dense
            abSingleFlat<Op,E><<<blocks, threads, 0, stream>>>(a, b, out, count);

        } else if (aDesc.isDense()) {
            // dense op dense --> dense
            abFlatFlat<Op,E><<<blocks, threads, 0, stream>>>(a, b, out, count);

        } else {
            // strided op dense --> dense
        }
    } else {
        if (aDesc.isSingle()) {
            // single op strided --> dense
        } else if (aDesc.isDense()) {
            // dense op strided --> dense
        } else {
            // strided op strided --> dense
        }
    }
}

//==============================================================================
// Swift importable C interface functions
//==============================================================================

void srtAddR1Float(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    combine<Add,1,float>(a, aDesc, b, bDesc, out, oDesc, stream);
}

void srtAddR2Float(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    combine<Add,2,float>(a, aDesc, b, bDesc, out, oDesc, stream);
}

void srtAddR3Float(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    combine<Add,3,float>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//==============================================================================

void srtAddR1Float16(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    combine<Add,1,__half>(a, aDesc, b, bDesc, out, oDesc, stream);
}

void srtAddR2Float16(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    combine<Add,2,__half>(a, aDesc, b, bDesc, out, oDesc, stream);
}

void srtAddR3Float16(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream
) {
    combine<Add,3,__half>(a, aDesc, b, bDesc, out, oDesc, stream);
}
