#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <stdio.h>
#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "elementOps.h"
#include "kernelHelpers.h"

//------------------------------------------------------------------------------
// add
template<typename T>
__global__ void add(const void *va, const void *vb, void *vc, unsigned count) {
    const T* a = (T*)va; const T* b = (T*)vb; T* c = (T*)vc;
    GRID_STRIDE_LOOP(i, count) {
        c[i] = a[i] + b[i];
    }
}

// add2_Float16
// TODO: Is there a way to do a __hadd8?? 
__global__ void add2_Float16(const void *va, const void *vb, void *vc, unsigned count) {
    const __half2 *a = (__half2 *)va;
    const __half2 *b = (__half2 *)vb;
    __half2 *c = (__half2 *)vc;

    GRID_STRIDE_LOOP(i, count) {
        c[i] = __hadd2(a[i], b[i]);
    }
}

// add4_Float32
__global__ void add4_Float32(const void *va, const void *vb, void *vc, unsigned count) {
    const float4 *a = (float4 *)va;
    const float4 *b = (float4 *)vb;
    float4 *c = (float4 *)vc;

    GRID_STRIDE_LOOP(i, count) { c[i] = a[i] + b[i]; }
}

//------------------------------------------------------------------------------
// Swift importable C functions
hipError_t srtAdd(hipDataType type, const void *a, const void *b, void *c,
                    unsigned count, hipStream_t stream) {
    KernelPreCheck(stream);
    unsigned blocks = BLOCK_COUNT(count);
    unsigned threads = THREADS_PER_BLOCK;
    switch(type) {
        case HIP_R_8I: add<char> <<<blocks, threads, 0, stream>>>(a, b, c, count); break;
        case HIP_R_8U: add<unsigned char> <<<blocks, threads, 0, stream>>>(a, b, c, count); break;
        case HIP_R_16I: add<short> <<<blocks, threads, 0, stream>>>(a, b, c, count); break;
        case HIP_R_16U: add<unsigned short> <<<blocks, threads, 0, stream>>>(a, b, c, count); break;
        case HIP_R_16F: {
            int elements = shiftDownRoundingUp(count, 1);
            add2_Float16<<<BLOCK_COUNT(elements), threads, 0, stream>>>(a, b, c, elements); 
            break;
        }

        case HIP_R_32F: {
            int elements = shiftDownRoundingUp(count, 2);
            add4_Float32<<<BLOCK_COUNT(elements), threads, 0, stream>>>(a, b, c, elements);
            break;
        }

        case HIP_R_64F: add<double> <<<blocks, threads, 0, stream>>>(a, b, c, count); break;
        default: printf("hipDataType not implemented"); assert(false);
    }
    return KernelPostCheck(stream);
}
