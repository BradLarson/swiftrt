#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <__clang_cuda_runtime_wrapper.h>
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "elementOps.h"
#include "kernelHelpers.h"

//==============================================================================
// vector ops
__device__ inline float4 operator +(const float4& l, const float4& r) {
    return make_float4(l.x + r.x, l.y + r.y, l.z + r.z, l.w + r.w);
}
  
__device__ inline double4 operator +(const double4& l, const double4& r) {
    return make_double4(l.x + r.x, l.y + r.y, l.z + r.z, l.w + r.w);
}

// #if (__CUDA_ARCH__ < 800)
// __device__ __forceinline__ __hip_bfloat162 operator+(const __hip_bfloat162& l, const __hip_bfloat162& r) {
//     __hip_bfloat162 c;
//     c.x = __float2bfloat16_rn(__bfloat162float(l.x) + __bfloat162float(r.x));
//     c.y = __float2bfloat16_rn(__bfloat162float(l.y) + __bfloat162float(r.y));
//     return c;
// }
// #endif

__device__ __forceinline__ __hip_bfloat162 add(const __hip_bfloat162& l, const __hip_bfloat162& r) {
    __hip_bfloat162 c;
    c.x = __float2bfloat16_rn(__bfloat162float(l.x) + __bfloat162float(r.x));
    c.y = __float2bfloat16_rn(__bfloat162float(l.y) + __bfloat162float(r.y));
    return c;
}

//------------------------------------------------------------------------------
// add
template<typename T>
__global__ void add(const void *va, const void *vb, void *vc, unsigned count) {
    const T* a = (T*)va; const T* b = (T*)vb; T* c = (T*)vc;
    GRID_STRIDE_LOOP(i, count) {
        c[i] = a[i] + b[i];
    }
}

template<typename T>
__global__ void add_bfloat162(const void *va, const void *vb, void *vc, unsigned count) {
    const T* a = (T*)va; const T* b = (T*)vb; T* c = (T*)vc;

    GRID_STRIDE_LOOP(i, count) {
        #if (__CUDA_ARCH__ >= 800)
            c[i] = a[i] + b[i];
        #else
            c[i] = add(a[i], b[i]);
        #endif
    }
}

//------------------------------------------------------------------------------
// Swift importable C functions
hipError_t srtAdd(
    hipDataType type, 
    const void *a,
    long countA, 
    const void *b, 
    long countB, 
    void *c,
    long countC, 
    hipStream_t stream
) {
    // make sure sizes fit within Cuda limitations
    assert(countA > 0 && countA <= INT32_MAX &&
        countB > 0 && countB <= INT32_MAX &&
        countC > 0 && countC <= INT32_MAX);

    KernelPreCheck(stream);
    unsigned blocks = BLOCK_COUNT(countC);
    unsigned threads = THREADS_PER_BLOCK;
    switch(type) {
        // case HIP_R_8I: add<char> <<<blocks, threads, 0, stream>>>(a, b, c, countC); break;
        // case HIP_R_8U: add<unsigned char> <<<blocks, threads, 0, stream>>>(a, b, c, countC); break;
        // case HIP_R_16I: add<short> <<<blocks, threads, 0, stream>>>(a, b, c, countC); break;
        // case HIP_R_16U: add<unsigned short> <<<blocks, threads, 0, stream>>>(a, b, c, countC); break;

        case HIP_R_16F: {
            int count = shiftDownRoundingUp(countC, 1);
            add<__half2><<<BLOCK_COUNT(count), threads, 0, stream>>>(a, b, c, count);
            break;
        }
        case HIP_R_16BF: {
            int count = shiftDownRoundingUp(countC, 1);
            add_bfloat162<__hip_bfloat162><<<BLOCK_COUNT(count), threads, 0, stream>>>(a, b, c, count);
            break;
        }
        case HIP_R_32F: {
            int count = shiftDownRoundingUp(countC, 2);
            add<float4><<<BLOCK_COUNT(count), threads, 0, stream>>>(a, b, c, count);
            break;
        }
        case HIP_R_64F: {
            int count = shiftDownRoundingUp(countC, 2);
            add<double4><<<BLOCK_COUNT(count), threads, 0, stream>>>(a, b, c, count);
            break;
        }
        default: printf("hipDataType not implemented"); assert(false);
    }
    return KernelPostCheck(stream);
}

//------------------------------------------------------------------------------
hipError_t srtAddStrided(
    hipDataType type,
    long dims,
    const void *a,
    const int* stridesA, 
    const void *b, 
    const int* stridesB, 
    void *c,
    const int* stridesC, 
    hipStream_t stream
) {
    return hipSuccess;
}
